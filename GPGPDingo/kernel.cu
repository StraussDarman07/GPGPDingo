#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cstdlib>
#include <math.h>
// Matrix is ROW Major in memory!

const int matWidth = 1024;
const int size = matWidth * matWidth;

void matMulHost(float* result, const float* a, const float* b, int size)
{
	for (int row = 0; row < size; row++)
	{
		for (int column = 0; column < size; column++)
		{
			result[column + row * size] = 0;
			for (int index = 0; index < size; index++)
			{
				result[column + row * size] += a[row * size + index] * b[column + size * index];
			}
		}


	}
}

__global__ void matMulCuda(float* result, const float* a, const float* b, int size)
{
	int row = blockIdx.x;
	int column = blockIdx.y;

	result[column + row * size] = 0;
	for (int index = 0; index < size; index++)
	{
		result[column + row * size] += a[row * size + index] * b[column + size * index];
	}
}

float getRandom()
{
	return ((float)rand()) / ((float)RAND_MAX);
}

#define CUDA_ERROR_CHECK(Value) if (Value != hipSuccess) {printf("cudafailed!"); return 1;};

int main()
{
	srand((unsigned)139213);
	// Init Random Seed
	/*std::default_random_engine randomEngine;
	std::uniform_real_distribution<float> distribution(0.f, 10.f);
	auto dice = std::bind(distribution, randomEngine);*/
	float *aCpu = (float*)malloc(sizeof(float) * size);
	for (int i = 0; i < size; ++i)
		*(aCpu + i) = getRandom();

	float *bCpu = (float*)malloc(sizeof(float) * size);
	for (int i = 0; i < size; ++i)
		*(bCpu + i) = getRandom();

	float *cpuResult = (float*)malloc(sizeof(float) * size);
	float *hostCudaResult = (float*)malloc(sizeof(float) * size);
	matMulHost(cpuResult, aCpu, bCpu, matWidth);


	float *cudaA, *cudaB, *cudaResult;
	CUDA_ERROR_CHECK(hipMalloc(&cudaA, sizeof(float) * size));
	CUDA_ERROR_CHECK(hipMalloc(&cudaB, sizeof(float) * size));
	CUDA_ERROR_CHECK(hipMalloc(&cudaResult, sizeof(float) * size));

	CUDA_ERROR_CHECK(hipMemcpy(cudaA, aCpu, size * sizeof(float), hipMemcpyHostToDevice));
	CUDA_ERROR_CHECK(hipMemcpy(cudaB, bCpu, size * sizeof(float), hipMemcpyHostToDevice));

	dim3 blockDim;
	blockDim.x = 1024;
	blockDim.y = 1024;

	matMulCuda << <blockDim, 1 >> >(cudaResult, cudaA, cudaB, matWidth);
	hipDeviceSynchronize();

	hipMemcpy(hostCudaResult, cudaResult, size * sizeof(float), hipMemcpyDeviceToHost);

	hipFree(cudaA);
	hipFree(cudaB);
	hipFree(cudaResult);

	for (int i = 0; i < size; ++i)
	{
		if (cpuResult[i] != hostCudaResult[i])
			printf("SHITY");
	}

	free(aCpu);
	free(bCpu);
	free(cpuResult);
	free(hostCudaResult);
	return 0;
}





