#include "hip/hip_runtime.h"
#include "math_functions.h"



__global__ void toOneChannel(unsigned char *data, int width, int height, int components)
{
	int column = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;

	if (row >= height || column >= width)
		return;

	unsigned char * threadData = data + (components * (column + row * width));

	for (int i = 0; i < components - 1; i++)
	{
		threadData[i] = 0;
	}
}

__global__ void toGrayScale(unsigned char *output, unsigned char *input, int width, int height, int components)
{
	int column = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;

	if (row >= height || column >= width)
		return;

	int index = column + row * width;
	unsigned char * threadData = input + components * index;
	unsigned char * outputData = output + index;

	const float partRed = 0.299f;
	const float partGreen = 0.587f;
	const float partBlue = 0.114;

	unsigned char greyScale = partBlue * threadData[0] + partGreen * threadData[1] + partRed * threadData[2];

	outputData[0] = greyScale;
}

__global__ void sobel(unsigned char *output, unsigned char *input, int width, int height)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (y >= height || x >= width)
		return;

	const int BLOCK_SIZE = 16;

	// Where does our data start
	int blockStartIndexX = blockIdx.x * blockDim.x - 1;
	int blockStartIndexY = blockIdx.y * blockDim.y - 1;

	// Clamp to edge
	if (blockStartIndexX < 0)
		blockStartIndexX = 0;

	if (blockStartIndexX >= width)
		blockStartIndexX = blockDim.x - 1;

	if (blockStartIndexY < 0)
		blockStartIndexY = 0;

	if (blockStartIndexY >= height)
		blockStartIndexY = blockDim.y - 1;

	// Shared Data
	__shared__ unsigned char pixels[BLOCK_SIZE + 2][BLOCK_SIZE + 2];

	// Where is our data
	unsigned char* cacheInput = input + (blockStartIndexX + blockStartIndexY * width);

	// Linear index (16x16 -> 0..255)
	int threadIndex = threadIdx.x + threadIdx.y * blockDim.x;

	int maxLoadSizeBytes = (BLOCK_SIZE + 2) * (BLOCK_SIZE + 2); // 18x18 Block -> 324 Bytes
	int maxIndexBytes = maxLoadSizeBytes / sizeof(short); // 18x18 Block -> Index 162

	if (threadIndex < maxIndexBytes)
	{
		// Calculate offset
		int offsetInBytes = threadIndex * sizeof(short);

		int block_half = (BLOCK_SIZE + 2) / 2;
		int byteRow = offsetInBytes / (BLOCK_SIZE + 2);
		int byteCol = threadIndex % block_half * 2;

		int offset = byteCol + byteRow * width;
		//int offsetBuffer = byteCol + byteRow * (BLOCK_SIZE + 2);

		// Copy Data
		unsigned char* toLoad = cacheInput + offset;
		/**(&pixels[0][0] + offsetBuffer) = *toLoad;
		*(&pixels[0][0] + offsetBuffer + 1) = *(toLoad + 1);*/

		pixels[byteRow][byteCol] = *toLoad;
		pixels[byteRow][byteCol + 1] = *(toLoad + 1);
	}

	__syncthreads();


	// Sobel weights
	float weightsX[9] = { -1, -2, -1,
						   0,  0,  0,
						   1,  2,  1 };

	float weightsY[9] = { -1,  0,  1,
						  -2,  0,  2,
						  -1,  0,  1 };

	int offsetY[9] = { -1,  -1,  -1,
						0,   0,   0,
						1,   1,   1 };

	int offsetX[9] = { -1,   0,   1,
					   -1,   0,   1,
					   -1,   0,   1 };


	float pointX = 0.f;
	float pointY = 0.f;

#pragma unroll
	for (int i = 0; i < 9; i++)
	{
		int indexX = threadIdx.x + 1 + offsetX[i];
		int indexY = threadIdx.y + 1 + offsetY[i];

		unsigned char pixel = pixels[indexY][indexX];
		pointX += pixel * weightsX[i];
		pointY += pixel * weightsY[i];
	}


	// Do Sobel here!
	int index = x + y * width;
	unsigned char * outputData = output + index;
	outputData[0] = sqrtf(pointX * pointX + pointY * pointY);
}


__global__ void sobelTex(unsigned char *output, hipTextureObject_t input, int width, int height)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (y >= height || x >= width)
		return;


	float weightsX[9] = { -1, -2, -1,
						   0,  0,  0,
						   1,  2,  1 };

	float weightsY[9] = { -1,  0,  1,
						  -2,  0,  2,
						  -1,  0,  1 };

	int offsetY[9] = { -1,  -1,  -1,
						0,   0,   0,
						1,   1,   1 };

	int offsetX[9] = { -1,   0,   1,
					   -1,   0,   1,
					   -1,   0,   1 };


	float pointX = 0.f;
	float pointY = 0.f;

#pragma unroll
	for (int i = 0; i < 9; i++)
	{
		int indexX = x + offsetX[i];
		int indexY = y + offsetY[i];

		float xf = (float)indexX / (float)width;
		float yf = (float)indexY / (float)height;

		unsigned char pixel = tex2D<unsigned char>(input, xf, yf);
		pointX += pixel * weightsX[i];
		pointY += pixel * weightsY[i];
	}

	output[y * width + x] = sqrtf(pointX * pointX + pointY * pointY);
}

__global__ void histogramGlobal(unsigned int* hist, unsigned char* input, int size, int stride)
{
	int index = blockIdx.x * blockDim.x * stride + threadIdx.x;

	if (index > size - 1)
		return;

	int i = 0;
	while (i < stride && index < size)
	{
		int pixel = input[index];
		atomicAdd(&(hist[pixel]), 1);
		index += stride;
		i++;
	}
}

__global__ void histogramPrivate(unsigned int* hist, unsigned char* input, int size, int stride)
{
	int index = blockIdx.x * blockDim.x * stride + threadIdx.x;

	if (index > size - 1)
		return;

	__shared__ unsigned int histo_private[256];

	histo_private[threadIdx.x * 8 + 0] = 0;
	histo_private[threadIdx.x * 8 + 1] = 0;
	histo_private[threadIdx.x * 8 + 2] = 0;
	histo_private[threadIdx.x * 8 + 3] = 0;
	histo_private[threadIdx.x * 8 + 4] = 0;
	histo_private[threadIdx.x * 8 + 5] = 0;
	histo_private[threadIdx.x * 8 + 6] = 0;
	histo_private[threadIdx.x * 8 + 7] = 0;

	__syncthreads();

	int i = 0;
	while (i < stride && index < size)
	{
		int pixel = input[index];
		atomicAdd(&(histo_private[pixel]), 1);
		index += stride;
		i++;
	}

	__syncthreads();

	atomicAdd(&(hist[threadIdx.x * 8 + 0]), histo_private[threadIdx.x * 8 + 0]);
	atomicAdd(&(hist[threadIdx.x * 8 + 1]), histo_private[threadIdx.x * 8 + 1]);
	atomicAdd(&(hist[threadIdx.x * 8 + 2]), histo_private[threadIdx.x * 8 + 2]);
	atomicAdd(&(hist[threadIdx.x * 8 + 3]), histo_private[threadIdx.x * 8 + 3]);
	atomicAdd(&(hist[threadIdx.x * 8 + 4]), histo_private[threadIdx.x * 8 + 4]);
	atomicAdd(&(hist[threadIdx.x * 8 + 5]), histo_private[threadIdx.x * 8 + 5]);
	atomicAdd(&(hist[threadIdx.x * 8 + 6]), histo_private[threadIdx.x * 8 + 6]);
	atomicAdd(&(hist[threadIdx.x * 8 + 7]), histo_private[threadIdx.x * 8 + 7]);
}