#include "hip/hip_runtime.h"
#include "math_functions.h"



__global__ void toOneChannel(unsigned char *data, int width, int height, int components)
{
    int column = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (row >= height || column >= width)
        return;

    unsigned char * threadData = data + (components * (column + row * width));

    for (int i = 0; i < components - 1; i++)
    {
        threadData[i] = 0;
    }
}

__global__ void toGrayScale(unsigned char *output, unsigned char *input, int width, int height, int components)
{
    int column = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (row >= height || column >= width)
        return;

    int index = column + row * width;
    unsigned char * threadData = input + components * index;
    unsigned char * outputData = output + index;

    const float partRed = 0.299f;
    const float partGreen = 0.587f;
    const float partBlue = 0.114;

    unsigned char greyScale = partBlue * threadData[0] + partGreen * threadData[1] + partRed * threadData[2];

    outputData[0] = greyScale;
}

__global__ void sobel(unsigned char *output, unsigned char *input, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (y >= height || x >= width)
        return;

    const int BLOCK_SIZE = 16;

    // Where does our data start
    int blockStartIndexX = blockIdx.x * blockDim.x - 1;
    int blockStartIndexY = blockIdx.y * blockDim.y - 1;

    // Clamp to edge
    if (blockStartIndexX < 0)
        blockStartIndexX = 0;

    if (blockStartIndexX >= width)
        blockStartIndexX = blockDim.x - 1;

    if (blockStartIndexY < 0)
        blockStartIndexY = 0;

    if (blockStartIndexY >= height)
        blockStartIndexY = blockDim.y - 1;

    // Shared Data
    __shared__ unsigned char pixels[BLOCK_SIZE + 2][BLOCK_SIZE + 2];

    // Where is our data
    unsigned char* cacheInput = input + (blockStartIndexX + blockStartIndexY * width);

    // Linear index (16x16 -> 0..255)
    int threadIndex = threadIdx.x + threadIdx.y * blockDim.x;

    int maxLoadSizeBytes = (BLOCK_SIZE + 2) * (BLOCK_SIZE + 2); // 18x18 Block -> 324 Bytes
    int maxIndexBytes = maxLoadSizeBytes / sizeof(short); // 18x18 Block -> Index 162

    if (threadIndex < maxIndexBytes)
    {
        // Calculate offset
        int offsetInBytes = threadIndex * sizeof(short);

        int block_half = (BLOCK_SIZE + 2) / 2;
        int byteRow = offsetInBytes / (BLOCK_SIZE + 2);
        int byteCol = threadIndex % block_half * 2;

        int offset = byteCol + byteRow * width;
        //int offsetBuffer = byteCol + byteRow * (BLOCK_SIZE + 2);

        // Copy Data
        unsigned char* toLoad = cacheInput + offset;
        /**(&pixels[0][0] + offsetBuffer) = *toLoad;
        *(&pixels[0][0] + offsetBuffer + 1) = *(toLoad + 1);*/

        pixels[byteRow][byteCol] = *toLoad;
        pixels[byteRow][byteCol + 1] = *(toLoad + 1);
    }

    __syncthreads();


    // Sobel weights
    float weightsX[9] = { -1, -2, -1,
                           0,  0,  0,
                           1,  2,  1 };

    float weightsY[9] = { -1,  0,  1,
                          -2,  0,  2,
                          -1,  0,  1 };

    int offsetY[9] = { -1,  -1,  -1,
                        0,   0,   0,
                        1,   1,   1 };

    int offsetX[9] = { -1,   0,   1,
                       -1,   0,   1,
                       -1,   0,   1 };


    float pointX = 0.f;
    float pointY = 0.f;

#pragma unroll
    for (int i = 0; i < 9; i++)
    {
        int indexX = threadIdx.x + 1 + offsetX[i];
        int indexY = threadIdx.y + 1 + offsetY[i];

        unsigned char pixel = pixels[indexY][indexX];
        pointX += pixel * weightsX[i];
        pointY += pixel * weightsY[i];
    }


    // Do Sobel here!
    int index = x + y * width;
    unsigned char * outputData = output + index;
    outputData[0] = sqrtf(pointX * pointX + pointY * pointY);
}


__global__ void sobelTex(unsigned char *output, hipTextureObject_t *input, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (y >= height || x >= width)
        return;

	unsigned int result = tex2D<unsigned int>(*input,x,y);
		//just that it is white
	unsigned char  test = (unsigned char) (int) result;
    output[y * width + x] = test;
}